/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.cu"

int main (int argc, char *argv[])
{
    //set standard seed
    srand(217);

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    float *A_d0, *B_d0, *C_d0;
    float *A_d1, *B_d1, *C_d1;
    size_t A_sz, B_sz, C_sz;
    unsigned VecSize;
   
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        VecSize = 1000000;

    } else if (argc == 2) {
      VecSize = atoi(argv[1]);   
    }
      else {
        printf("\nOh no!\nUsage: ./vecAdd <Size>");
        exit(0);
    }

    A_sz = VecSize;
    B_sz = VecSize;
    C_sz = VecSize;
    //TODO: modify to use hipHostAlloc instead of malloc
    /*
    A_h = (float*) malloc( sizeof(float)*A_sz );
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    B_h = (float*) malloc( sizeof(float)*B_sz );
    for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    C_h = (float*) malloc( sizeof(float)*C_sz );
    */
    hipHostAlloc(&A_h, sizeof(float)*A_sz, hipHostMallocDefault);
    hipHostAlloc(&B_h, sizeof(float)*B_sz, hipHostMallocDefault);
    hipHostAlloc(&C_h, sizeof(float)*C_sz, hipHostMallocDefault);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    size Of vector: %u x %u\n  ", VecSize);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

		//INSERT CODE HERE
		//create streams first before allocating memory
		hipStream_t stream0, stream1;
		hipStreamCreate(&stream0); 
		hipStreamCreate(&stream1); 
    //allocate memory for vectors A, B, and C on the device.
    //also error check for each malloc
    hipError_t err_A =  hipMalloc((void**)&A_d0, A_sz/2*sizeof(float));
    if (err_A != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_A), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }
   	err_A =  hipMalloc((void**)&A_d1, A_sz/2*sizeof(float));
    if (err_A != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_A), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }
    hipError_t err_B = hipMalloc((void**)&B_d0, B_sz/2*sizeof(float));
    if (err_B != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_B), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }
    err_B = hipMalloc((void**)&B_d1, B_sz/2*sizeof(float));
    if (err_B != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_B), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }
    hipError_t err_C = hipMalloc((void**)&C_d0, C_sz/2*sizeof(float));
    if (err_C != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_C), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }
    err_C = hipMalloc((void**)&C_d1, C_sz/2*sizeof(float));
    if (err_C != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_C), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    //only copy A and B from host to device
    //vector sum result will be in C, which is copied from device to host
    //hipMemcpy(A_d, A_h, A_sz*sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(B_d, B_h, B_sz*sizeof(float), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel  ---------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
    /*** basic non-stream kernel call ***/
    ///basicVecAdd(A_d, B_d, C_d, A_h, B_h, C_h, VecSize); //In kernel.cu

    /***muli-stream code***/
    int segSize = VecSize/2+1;
    for (int i = 0; i < VecSize; i+= segSize*2) {
    	hipMemcpyAsync(A_d0, A_h+i, segSize*sizeof(float), hipMemcpyHostToDevice, stream0);
    	hipMemcpyAsync(B_d0, B_h+i, segSize*sizeof(float), hipMemcpyHostToDevice, stream0);
    	hipMemcpyAsync(A_d1, A_h+i+segSize, segSize*sizeof(float), hipMemcpyHostToDevice, stream1);
    	hipMemcpyAsync(B_d1, B_h+i+segSize, segSize*sizeof(float), hipMemcpyHostToDevice, stream1);

    	VecAdd<<<segSize/256 + 1, 256, 0, stream0>>>(segSize, A_d0, B_d0, C_d0); 
    	VecAdd<<<segSize/256 + 1, 256, 0, stream1>>>(segSize, A_d1, B_d1, C_d1); 
			
			hipMemcpyAsync(C_h+i, C_d0, segSize*sizeof(float), hipMemcpyDeviceToHost, stream0);
			hipMemcpyAsync(C_h+i+segSize, C_d0, segSize*sizeof(float), hipMemcpyDeviceToHost, stream1);
		}

    cuda_ret = hipDeviceSynchronize();
		if(cuda_ret != hipSuccess) {
    	printf("CUDA failure %s:%d: '%s'\n", __FILE__,__LINE__,
    			hipGetErrorString(cuda_ret)); 
			FATAL("Unable to launch/execute kernel");
		}
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    //copy C (result) back to host
    //hipMemcpy(C_h, C_d, C_sz*sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, VecSize);


    // Free memory ------------------------------------------------------------

		//TODO: modify to use hipHostFree
    hipHostFree(A_h);
    hipHostFree(B_h);
    hipHostFree(C_h);

    //INSERT CODE HERE
    //free memory allocated in the device
    //also error check for each hipFree
    err_A = hipFree(A_d0);
    if (err_A != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_A), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }
    err_A = hipFree(A_d1);
    if (err_A != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_A), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }
    err_B = hipFree(B_d0);
    if (err_B != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_B), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }
    err_B = hipFree(B_d1);
    if (err_B != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_B), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }
    err_C = hipFree(C_d0);
    if (err_C != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_C), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }
    err_C = hipFree(C_d1);
    if (err_C != hipSuccess) {
    	printf("%s in %s at line %d\n", hipGetErrorString(err_C), __FILE__, __LINE__);
    	exit(EXIT_FAILURE);
    }
    return 0;

}
