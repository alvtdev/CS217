/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE

  	__shared__ float ds_A[TILE_SIZE][TILE_SIZE];
  	__shared__ float ds_B[TILE_SIZE][TILE_SIZE];

  	//declare and initialize ids
  	int bx = blockIdx.x;
  	int by = blockIdx.y;
  	int tx = threadIdx.x;
  	int ty = threadIdx.y;

		//calculate row/column
		int row = blockDim.y * by + ty; 
		int col = blockDim.x * bx + tx;
		
		//declare P value variable for matrix sum
		float pValue = 0.0;

		//compute the P element by looping over M and N matrices
		for (int p = 0; p < ((k-1)/TILE_SIZE+1) ; p++) {
			//load A and B tiles into shared memory
			//loads 0 into memory if out of bounds
			if(row < m && (p*TILE_SIZE + tx) < k) {
				ds_A[ty][tx] = A[row*k + p*TILE_SIZE + tx];
			}
			else {
				ds_A[ty][tx] = 0.0;
			}
			if (col < n && (p*TILE_SIZE + ty) < k) {
				//ds_B[ty][tx] = B[(p*TILE_SIZE*+ty)*n + col];
			}
			else {
				ds_B[ty][tx] = 0.0;
			}
			__syncthreads();

			//compute p value 
			if (row < m && col < n) {
				for (int j = 0; j < TILE_SIZE; j++) {
					pValue += ds_A[ty][j] * ds_B[j][tx];
				}
			}
			__syncthreads();
		}

		//assign pValues to C if within bounds
		if(row < m && col < n) {
			C[row*n+col] = pValue;
		}

}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------
		//printf("INIT THREADS AND KERNEL GRIDS...\n"); fflush(stdout);

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    //INSERT CODE HERE
		dim3 dimGrid(((n-1)/TILE_SIZE)+1, ((m-1)/TILE_SIZE)+1);

		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);



    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE

    mysgemm<<<dimGrid, dimBlock>>>(m, n, k, A, B, C);

}


